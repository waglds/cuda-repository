#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>

#include "calc.h"

/*
 *s
 * Function Declaration
 *
 * */
__host__ void calc(int argc, char** argv);
__host__ void cpu_function(double*, int*);
__global__ void device_function_main(int*, int*, int*, float, bool);

/*
 * Main Function
 * */
int main(int argc, char ** argv) {
	
	calc(argc, argv);
	getchar();
	return 0;
}
